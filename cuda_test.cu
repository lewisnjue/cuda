
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function
__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int a = 5, b = 7, c;
    int *d_c;

    // Allocate memory on GPU
    hipMalloc((void **)&d_c, sizeof(int));

    // Launch kernel with 1 block and 1 thread
    add<<<1, 1>>>(a, b, d_c);

    // Copy result from GPU to CPU
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result from GPU: " << c << std::endl;

    // Free allocated memory
    hipFree(d_c);

    return 0;
}
